
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include <sys/time.h>
#define imin(a,b) (a<b?a:b)

const int N = 16777216;
const int TH_B = 512;
const int blocksPerGrid = imin( 32, (N+TH_B-1) / TH_B );


long long start_timer() {

struct timeval tv;

gettimeofday(&tv, NULL);

return tv.tv_sec * 1000000 + tv.tv_usec;

}


long long stop_timer(long long start_time,char *name) {

struct timeval tv;

gettimeofday(&tv, NULL);

long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;

float total_time = (end_time - start_time)/1000000.0;

printf("Value of Blocks Per Grid : %d",blocksPerGrid);

//print execution time for cpu
if(name=="Cpu")
{

printf("\nC) (T%s) Execution Time for Serial Algorithm or %s : %.5f sec\n",name,name,total_time);
}

//print execution time for gpu and kernel time
if(name=="Gpu")
{
printf("\nE) Kernel execution Time is %.5f sec\n",total_time);
printf("\nF) (T%s) Execution Time for Parallel Algorithm or %s :  %.5f sec\n",name,name,total_time);
}

//print execution time for memory allocation in gpu
if(name=="memalloctgpu")
{

printf("\nB) Memory allocation Time for GPU is : %.5f sec\n",total_time);

}

//print execution time for memory allocation in cpu
if(name=="memalloctcpu")
{

printf("\nA) Memory allocation Time for CPU is : %.5f sec\n",total_time);

}


//print condition for cpu to gpu time

if(name=="c2g")
{
printf("\nD) Data Transfer from CPU to GPU time is : %.5f sec\n",total_time);

}
//print condition for gpu to cpu transfer time
if(name=="g2c")
{

printf("\nG) Data Transfer from GPU to CPU time is : %.5f sec\n",total_time);

}

return ((end_time) - (start_time));

}


__global__ void GPU_big_dot( float *a, float *b, float *c ) {
    __shared__ float cache[TH_B];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float   temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // assign the cache 
    cache[cacheIndex] = temp;

    // synchronize threads in this block
    __syncthreads();

    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];


}

float CPU_big_dot(float *a,float *b)
{
              float  cpu_sum;
              for(int tid =0;tid<N;tid++)
              {
                      a[tid]=tid;
                      b[tid]= tid * 2;
                      cpu_sum += a[tid] * b[tid];
               }
              return cpu_sum;
}


int main( void ) {

   long long s_t;
   long long s_tt;
   long long s_tt_g2c;
   long long s_t_c2g;
   long long cpu_i;
   long long gpu_i;
   float spu;
   float CPU_SUM;
   long long s_t_cpu_memalloc;
   long long s_t_gpu_memalloc;


    float   *a, *b, c, *partial_c;
    float   *d_a, *d_b, *d_partial_c;

    s_t_cpu_memalloc = start_timer();

    // allocate memory on the cpu side
    a = (float*)malloc( N*sizeof(float) );
    b = (float*)malloc( N*sizeof(float) );
    partial_c = (float*)malloc( blocksPerGrid*sizeof(float) );
    
    stop_timer(s_t_cpu_memalloc,(char*)"memalloctcpu");

    s_t_gpu_memalloc = start_timer();


    // allocate the memory on the GPU
    hipMalloc( (void**)&d_a,
                              N*sizeof(float) ) ;
    hipMalloc( (void**)&d_b,
                              N*sizeof(float) ) ;
    hipMalloc( (void**)&d_partial_c,
                              blocksPerGrid*sizeof(float) ) ;

    stop_timer(s_t_gpu_memalloc,(char*)"memalloctgpu");


    //fill in the host memory with data
    for (int i=0; i<N; i++) {
        a[i] = i;
        //b[i] = i * 2;
         b[i] = i;
    }

   s_t = start_timer();
    CPU_SUM = CPU_big_dot(a,b);
    cpu_i=stop_timer(s_t,(char*)"Cpu");
 

    s_t_c2g = start_timer();
     
    
    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( d_a, a, N*sizeof(float),
                              hipMemcpyHostToDevice ) ;
    hipMemcpy( d_b, b, N*sizeof(float),
                              hipMemcpyHostToDevice ) ;

    stop_timer(s_t_c2g,(char*)"c2g");

  s_tt = start_timer();
  GPU_big_dot<<<blocksPerGrid,TH_B>>>( d_a, d_b,
                                            d_partial_c );


  gpu_i=stop_timer(s_tt, (char*)"Gpu");
    // copy the array 'c' back from the GPU to the CPU

   s_tt_g2c = start_timer();
   
    hipMemcpy( partial_c, d_partial_c,
                              blocksPerGrid*sizeof(float),
                              hipMemcpyDeviceToHost ) ;
   stop_timer(s_tt_g2c, (char*)"g2c");



  spu=(float)((float)cpu_i/(float)gpu_i);
  printf("\nH) Total SpeedUp is : %f \n",spu);

    // finish up on the CPU side
                                        
   c = 0;
    for (int i=0; i<blocksPerGrid; i++) {
        c += partial_c[i];
    }

    printf( "\nI) GPU dot-product value is %f = %.6g\n", c,c);


    printf( "\nJ) CPU dot-product value is %f = %.6g\n\n", CPU_SUM,CPU_SUM );


    // free memory on the gpu side
    hipFree( d_a ) ;
    hipFree( d_b ) ;
    hipFree( d_partial_c ) ;

    // free memory on the cpu side
    free( a );
    free( b );
    free( partial_c );
}

